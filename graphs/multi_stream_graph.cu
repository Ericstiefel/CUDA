#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_CHECK(call)                                                 \
do {                                                                     \
    hipError_t err = call;                                              \
    if (err != hipSuccess) {                                            \
        fprintf(stderr, "Cuda Error in file %s at line %d: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err));                                \
        exit(EXIT_FAILURE);                                              \
    }                                                                    \
} while (0)

__global__ void kernelA(const float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = input[idx] * 5.0f;
    }
}

__global__ void kernelB(const float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = input[idx] * 4.0f;
    }
}

__global__ void kernelC(const float* inputA, const float* inputB, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = inputA[idx] + inputB[idx];
    }
}

int main() {
    int N = 1000;
    size_t size = N * sizeof(float);

    float *h_ainp, *h_aout, *h_binp, *h_bout, *h_cout;
    float *d_ainp, *d_aout, *d_binp, *d_bout, *d_cout;
    
    CUDA_CHECK(hipHostMalloc((void**)&h_ainp, size));
    CUDA_CHECK(hipHostMalloc((void**)&h_aout, size));

    CUDA_CHECK(hipHostMalloc((void**)&h_binp, size));
    CUDA_CHECK(hipHostMalloc((void**)&h_bout, size));

    CUDA_CHECK(hipHostMalloc((void**)&h_cout, size));

    for (int i = 0; i < N; ++i) { h_ainp[i] = (float)i; }
    for (int i = 0; i < N; ++i) { h_binp[i] = (float)(i % 25); }

    CUDA_CHECK(hipMalloc((void**)&d_ainp, size));
    CUDA_CHECK(hipMalloc((void**)&d_aout, size));

    CUDA_CHECK(hipMalloc((void**)&d_binp, size));
    CUDA_CHECK(hipMalloc((void**)&d_bout, size));

    CUDA_CHECK(hipMalloc((void**)&d_cout, size));

    hipGraph_t graph;
    hipGraphExec_t graph_executable;

    CUDA_CHECK(hipGraphCreate(&graph));

    hipStream_t streamA, streamB, streamC;

    hipEvent_t kerA, kerB;


    CUDA_CHECK(hipStreamCreate(&streamA));
    CUDA_CHECK(hipStreamCreate(&streamB));
    CUDA_CHECK(hipStreamCreate(&streamC));

    CUDA_CHECK(hipEventCreate(&kerA));
    CUDA_CHECK(hipEventCreate(&kerB));

    CUDA_CHECK(hipStreamBeginCapture(streamA, hipStreamCaptureModeGlobal));

    CUDA_CHECK(cudaMemcpyAsyc(d_ainp, h_ainp, size, hipMemcpyHostToDevice, streamA));

    CUDA_CHECK(cudaMemcpyAsyc(d_binp, h_binp, size, hipMemcpyHostToDevice, streamB));

    CUDA_CHECK(kernelA<<<1, N, 0, streamA>>>(d_ainp, d_aout, N));
    CUDA_CHECK(kernelB<<<1, N, 0, streamB>>>(d_binp, d_bout, N));

    CUDA_CHECK(hipEventRecord(kerA, streamA));
    CUDA_CHECK(hipEventRecord(kerB, streamB));

    CUDA_CHECK(hipStreamWaitEvent(streamC, kerA));
    CUDA_CHECK(hipStreamWaitEvent(streamC, kerB));

    CUDA_CHECK(kernelC<<<1, N, 0, streamC>>>(d_aout, d_bout, d_cout, N));

    CUDA_CHECK(hipMemcpyAsync(h_cout, d_cout, size, hipMemcpyDeviceToHost, streamC));

    CUDA_CHECK(hipStreamEndCapture(streamA, &graph));

    CUDA_CHECK(hipGraphInstantiate(&graph_executable, graph));

    CUDA_CHECK(hipGraphLaunch(graph_executable, streamC));
    CUDA_CHECK(hipStreamSynchronize(streamC));

    CUDA_CHECK(cudaHostFree(h_ainp));
    // Rest of freeing memory

    return 0;
}