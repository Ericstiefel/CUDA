#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <studio.h>

#define CUDA_CHECK(call) \

do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "Cuda Error in file %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE) \
    } \
} while (0)


__global__ void kernelA(float* input, float* output, int N) {
    if (threadIdx.x < N) {
        output[threadIdx.x] = input[threadIdx.x] * 5.0f;
    }
}
__global__ void kernelB(float* input, float* output, int N) {
    if (threadIdx.x < N) {
        output[threadIdx.x] = input[threadIdx.x] * 4.0f;
    }
}
__global__ void kernelC(const float* inputA, const float* inputB, float* output, int N) {
    if (threadIdx.x < N) {
        output[threadIdx.x] = inputA[threadIdx.x] + inputB[threadIdx.x];
    }
}


void construct_graph() {
    hipGraph_t graph;
    hipGraphCreate(&graph);

    hipGraphNode_t cpyA, cpyB, cpyC, kerA, kerB, kerC;

    hipKernelNodeParams kerAP, kerBP, kerCP;
    cudaMemcpy3DParams memparA, memparB, memparC;


    // Assume all the params were defined here



    CUDA_CHECK(hipGraphAddMemcpyNode(&cpyA, graph, NULL, 0, &memparA));
    CUDA_CHECK(hipGraphAddMemcpyNode(&cpyB, graph, NULL, 0, &memparB));

    CUDA_CHECK(hipGraphAddKernelNode(&kerA, graph, &cpyA, 1, &kerAP));
    CUDA_CHECK(hipGraphAddKernelNode(&kerB, graph, &cpyB, 1, &kerAB));

    hipGraphNode_t c_dep[] = {kerA, kerB};

    CUDA_CHECK(hipGraphAddKernelNode(&kerC, graph, &c_dep, 2, &kerCP));

    hipGraphExec_t graph_executable;

    CUDA_CHECK(hipGraphInstantiate(&graph_executable, graph));

}