#include "hip/hip_runtime.h"
/*
Simple Graph using Stream Capture that multiplies every element by two, single stream.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(call)\
do {\
    hipError_t err = call;\

    if (err != hipSuccess) {\
        fprintf(stderr, "Cuda Failure in File %s on line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));\
        exit(EXIT_FAILURE);\
    }\
} while (0)

__global__ void ker(float* inp, float* out, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx < N) {
        out[idx] = inp[idx] * 2.0f;
    }
}

int main() {
    int N = 1000;

    size_t size = sizeof(float) * N;

    float *h_inp, *h_out;

    CUDA_CHECK(hipHostMalloc((void**)&h_inp, size));
    CUDA_CHECK(hipHostMalloc((void**)&h_out, size));

    for (int i = 0; i < N; ++i) {
        h_inp[i] = (float)i;
    }

    float *d_inp, *d_out;

    CUDA_CHECK(hipMalloc((void**)&d_inp, size));
    CUDA_CHECK(hipMalloc((void**)&d_out, size));


    hipGraph_t graph;
    hipGraphExec_t graph_executable;


    hipStream_t stream;

    hipStreamCreate(&stream);

    CUDA_CHECK(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    CUDA_CHECK(hipMemcpyAsync(d_inp, h_inp, size, hipMemcpyHostToDevice, stream));

    CUDA_CHECK(ker<<<1, N, 0, stream>>>(d_inp, d_out, N));

    CUDA_CHECK(hipMemcpyAsync(h_out, d_out, size, hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamEndCapture(stream, &graph));

    CUDA_CHECK(hipGraphInstantiate(&graph_executable, graph));


    printf("Launching graph...\n");
    CUDA_CHECK(hipGraphLaunch(graph_executable, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));
    printf("Graph execution complete.\n");

    printf("Result verification for index 10: %f (Expected: 20.0)\n", h_out[10]);

    CUDA_CHECK(hipGraphDestroy(graph));
    CUDA_CHECK(hipGraphExecDestroy(graph_executable));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipHostFree(h_inp));
    CUDA_CHECK(hipHostFree(h_out));
    CUDA_CHECK(hipFree(d_inp));
    CUDA_CHECK(hipFree(d_out));

    return 0;

}