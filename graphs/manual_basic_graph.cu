#include "hip/hip_runtime.h"
/*
Manual Graph implementation of the basic_graph.cu
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(call)\
do {\
    hipError_t err = call;\
    if (err != hipSuccess) {\
        fprintf(stderr, "Cuda Error in file %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));\
        exit(EXIT_FAILURE);\
    }\
} while (0)

__global__ void ker(float* inp, float* out, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N) {
        out[idx] = inp[idx] * 2.0f;
    }
}

int main() {
    int N = 1000;
    size_t size = N * sizeof(float);
    float *h_inp, *h_out, *d_inp, *d_out;

    CUDA_CHECK(hipHostMalloc((void**)&h_inp, size));
    CUDA_CHECK(hipHostMalloc((void**)&h_out, size));

    CUDA_CHECK(hipMalloc((void**)d_inp, size));
    CUDA_CHECK(hipMalloc((void**)d_out, size));

    hipGraph_t graph;
    hipGraphExec_t graph_executable;

    hipGraphCreate(&graph);

    hipGraphNode_t to_device, from_device, ker;

    // --- Fill params for the Host-to-Device copy ---
    hipMemcpy3DParms to_d_p = {0}; // Initialize to zero
    to_d_p.srcPtr   = make_hipPitchedPtr(h_inp, size, N, 1);
    to_d_p.dstPtr   = make_hipPitchedPtr(d_inp, size, N, 1);
    to_d_p.extent   = make_hipExtent(size, 1, 1);
    to_d_p.kind     = hipMemcpyHostToDevice;

    // --- Fill params for the Device-to-Host copy ---
    hipMemcpy3DParms from_d_p = {0};
    from_d_p.srcPtr   = make_hipPitchedPtr(d_out, size, N, 1);
    from_d_p.dstPtr   = make_hipPitchedPtr(h_out, size, N, 1);
    from_d_p.extent   = make_hipExtent(size, 1, 1);
    from_d_p.kind     = hipMemcpyDeviceToHost;

    void* kernel_args[] = {&d_inp, &d_out, &N};

    // --- Fill params for the kernel launch ---
    hipKernelNodeParams ker_params = {0};
    ker_params.func           = (void*)ker;
    ker_params.gridDim        = 1;
    ker_params.blockDim       = N;
    ker_params.sharedMemBytes = 0;
    ker_params.kernelParams   = kernel_args;
    ker_params.extra          = NULL;

    CUDA_CHECK(hipGraphAddMemcpyNode(&to_device, graph, NULL, 0, &to_d_p));
    CUDA_CHECK(hipGraphAddKernelNode(&ker, graph, to_device, 1, &ker_params));
    CUDA_CHECK(hipGraphAddMemcpyNode(&from_device, graph, ker, 1, &from_d_p));

    CUDA_CHECK(hipGraphInstantiate(&graph_executable, graph));


    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    CUDA_CHECK(hipGraphLaunch(graph_executable, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipGraphDestroy(graph));
    CUDA_CHECK(hipGraphExecDestroy(graph_executable));
    CUDA_CHECK(hipHostFree(h_inp));
    CUDA_CHECK(hipHostFree(h_out));
    CUDA_CHECK(hipFree(d_inp));
    CUDA_CHECK(hipFree(d_out));
    CUDA_CHECK(hipStreamDestroy(stream));


    return 0;
}