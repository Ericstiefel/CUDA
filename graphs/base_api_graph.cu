#include "hip/hip_runtime.h"
/*
This file will contain using cuda api to create a basic reusable graph
*/

#include <hip/hip_runtime.h>
#include <studio.h>

#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \

    if (err != hipSuccess) { \
        fprintf(stderr, "Cuda Error in file %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)


__global__ void kernelA(float* input, float* output, int N) {
    if (threadIdx.x < N) {
        output[threadIdx.x] = input[threadIdx.x] * 5.0f;
    }
}
__global__ void kernelB(float* input, float* output, int N) {
    if (threadIdx.x < N) {
        output[threadIdx.x] = input[threadIdx.x] * 4.0f;
    }
}
__global__ void kernelC(const float* inputA, const float* inputB, float* output, int N) {
    if (threadIdx.x < N) {
        output[threadIdx.x] = inputA[threadIdx.x] + inputB[threadIdx.x];
    }
}

int main() {
    int N = 1000;
    size_t size = N * sizeof(float);

    // Host memory
    float *h_inpA, *h_inpB, *h_outC;
    CUDA_CHECK(hipHostMalloc((void**)&h_inpA, size));
    CUDA_CHECK(hipHostMalloc((void**)&h_inpB, size));
    CUDA_CHECK(hipHostMalloc((void**)&h_outC, size));

    // Initialize host inputs
    for (int i = 0; i < N; ++i) { h_inpA[i] = (float)i; }
    for (int i = 0; i < N; ++i) { h_inpB[i] = (float)i * 2.0f; }

    // Device memory
    float *d_inpA, *d_inpB, *d_outA, *d_outB, *d_outC;
    CUDA_CHECK(hipMalloc((void**)&d_inpA, size));
    CUDA_CHECK(hipMalloc((void**)&d_inpB, size));
    CUDA_CHECK(hipMalloc((void**)&d_outA, size));
    CUDA_CHECK(hipMalloc((void**)&d_outB, size));
    CUDA_CHECK(hipMalloc((void**)&d_outC, size)); 

    hipStream_t streamA, streamB, streamC;
    CUDA_CHECK(hipStreamCreate(&streamA));
    CUDA_CHECK(hipStreamCreate(&streamB));
    CUDA_CHECK(hipStreamCreate(&streamC));

    hipEvent_t eventA_done, eventB_done;
    CUDA_CHECK(hipEventCreate(&eventA_done));
    CUDA_CHECK(hipEventCreate(&eventB_done));

    hipGraph_t graph;
    hipGraphExec_t graph_executable;

    CUDA_CHECK(hipStreamBeginCapture(streamA, hipStreamCaptureModeGlobal));


    CUDA_CHECK(hipMemcpyAsync(d_inpA, h_inpA, size, hipMemcpyHostToDevice, streamA));
    CUDA_CHECK(hipMemcpyAsync(d_inpB, h_inpB, size, hipMemcpyHostToDevice, streamB));


    kernelA<<<1, N, streamA>>>(d_inpA, d_outA, N);
    kernelB<<<1, N, streamB>>>(d_inpB, d_outB, N);

    CUDA_CHECK(hipEventRecord(eventA_done, streamA));
    CUDA_CHECK(hipEventRecord(eventB_done, streamB));

    CUDA_CHECK(hipStreamWaitEvent(streamC, eventA_done));
    CUDA_CHECK(hipStreamWaitEvent(streamC, eventB_done));

    kernelC<<<1, N, streamC>>>(d_outA, d_outB, d_outC, N);

    CUDA_CHECK(hipStreamEndCapture(streamA, &graph));

    CUDA_CHECK(hipGraphInstantiate(&graph_executable, graph));
    

    CUDA_CHECK(hipGraphLaunch(graph_executable, streamA)); // Reuse later

    CUDA_CHECK(hipMemcpyAsync(h_outC, d_outC, size, hipMemcpyDeviceToHost, streamC));

    CUDA_CHECK(hipStreamSynchronize(streamC)); // Result is now in h_outC


    CUDA_CHECK(hipGraphDestroy(graph));
    CUDA_CHECK(hipGraphExecDestroy(graph_executable));
    CUDA_CHECK(hipHostFree(h_inpA));
    CUDA_CHECK(hipHostFree(h_inpB));
    CUDA_CHECK(hipHostFree(h_outC));
    CUDA_CHECK(hipFree(d_inpA));
    CUDA_CHECK(hipFree(d_inpB));
    CUDA_CHECK(hipFree(d_outA));
    CUDA_CHECK(hipFree(d_outB));
    CUDA_CHECK(hipFree(d_outC));
    CUDA_CHECK(hipEventDestroy(eventA_done));
    CUDA_CHECK(hipEventDestroy(eventB_done));
    CUDA_CHECK(hipStreamDestroy(streamA));
    CUDA_CHECK(hipStreamDestroy(streamB));
    CUDA_CHECK(hipStreamDestroy(streamC));

    return 0;
}