#include "hip/hip_runtime.h"
/*
Gameplan:
Each block will compute one row of the input matrix.
This means we launch N blocks.

threadsPerBlock = TILE_SIZE (each thread will load one element from global to shared memory & do computations)

# of tiles iterated over = (N + TILE_SIZE - 1) / TILE_SIZE

Device called kernel: Find max element and normalized sum through reduction (O(log(n)) steps instead of O(n)). Two separate kernels.

local max and sum written to registers 

*/

#include <hip/hip_runtime.h>
#include <cmath>

#define TILE_SIZE 64

__device__ void block_reduce(float* s_data, float& val, bool is_max) {
    s_data[threadIdx.x] = val;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            if (is_max) { // No thread divergence because all threads in a warp execute the same path
                // Max reduction
                if (s_data[threadIdx.x + stride] > s_data[threadIdx.x]) {
                    s_data[threadIdx.x] = s_data[threadIdx.x + stride];
                }
            } else {
                // Sum reduction
                s_data[threadIdx.x] += s_data[threadIdx.x + stride];
            }
        }
        __syncthreads();
    }]
    val = s_data[0];
}

__global__ void softmax_fused_kernel(const float* __restrict__ input,
                                     float* __restrict__ output,
                                     int M, int N) {
    

    __shared__ float s_tile[TILE_SIZE];

    int row = blockIdx.x;
    int row_start_idx = row * N;
    int num_tiles = (N + TILE_SIZE - 1) / TILE_SIZE;

    float row_max = -FLT_MAX;
    for (int tile = 0; tile < num_tiles; ++tile) {
        int idx = tile * TILE_SIZE + threadIdx.x;
        
        float thread_val = (idx < N) ? input[row_start_idx + idx] : -FLT_MAX; // Padding not necessary (warp threads are placed in the 32 separate Shared Memory banks)

        block_reduce(s_tile, thread_val, true);
        
        if (threadIdx.x == 0) {
            if (thread_val > row_max) {
                row_max = thread_val;
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        s_tile[0] = row_max;
    }
    __syncthreads();
    row_max = s_tile[0];

    float row_sum = 0.0f;
    for (int tile = 0; tile < num_tiles; ++tile) {
        int idx = tile * TILE_SIZE + threadIdx.x;

        float thread_val = 0.0f;
        if (idx < N) {
            thread_val = expf(input[row_start_idx + idx] - row_max);
        }

        block_reduce(s_tile, thread_val, false);

        if (threadIdx.x == 0) {
            row_sum += thread_val;
        }
        __syncthreads();
    }
    
    if (threadIdx.x == 0) {
        s_tile[0] = row_sum;
    }
    __syncthreads();
    row_sum = s_tile[0];
    

    for (int tile = 0; tile < num_tiles; ++tile) {
        int idx = tile * TILE_SIZE + threadIdx.x;
        if (idx < N) {
            float val = expf(input[row_start_idx + idx] - row_max);
            output[row_start_idx + idx] = val / row_sum;
        }
    }
}

// Host launcher function
void softmax_launcher(float* d_input, float* d_output, int M, int N) {
    dim3 threadsPerBlock(TILE_SIZE);
    dim3 blocksPerGrid(M);

    softmax_fused_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, M, N);
}
