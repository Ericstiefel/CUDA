#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <ctime>
#include <cmath>
#include <cuda/pipeline>

// Reminder for later: Convert tile loads from DRAM to Asynch

using namespace nvcuda;

// ------------------ User-tunable model dims ------------------
#define SEQ_LEN 32      // number of tokens (M)
#define D_MODEL 128     // embedding dim (input feature size)
#define D_K 64          // query/key dim
#define D_V 64          // value dim

// WMMA tile params 
#define TILE_WIDTH 16
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

#define CEIL_DIV(x,y) (((x)+(y)-1)/(y))

#define CUDA_CHECK(call)                                                       \
do {                                                                           \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
        fprintf(stderr, "CUDA Error in file %s at line %d: %s\n", __FILE__,    \
                __LINE__, hipGetErrorString(err));                            \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)


__global__ void init_random_matrix(half* ptr, int rows, int cols, unsigned long seed) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < rows && col < cols) {
        int idx = row * cols + col;
        hiprandState localState;
        hiprand_init(seed, idx, 0, &localState);
        float rv = hiprand_uniform(&localState);
        ptr[idx] = __float2half(rv);
    }
}


__global__ void matmul(const half* __restrict__ A,
                       const half* __restrict__ B,
                       half* __restrict__ C,
                       const int M,
                       const int N,
                       const int K)
{
    int global_row = blockIdx.y * blockDim.y + threadIdx.y;
    int global_col = blockIdx.x * blockDim.x + threadIdx.x;

    int warp_m = global_row / WMMA_M;
    int warp_n = global_col / WMMA_N;

    // shared buffers (3-stage)
    __shared__ half A_tile[3][WMMA_M][WMMA_K];
    __shared__ half B_tile[3][WMMA_K][WMMA_N];

    wmma::fragment<matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> A_frag[3];
    wmma::fragment<matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> B_frag[3];

    wmma::fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, float> C_frag;
    wmma::fill_fragment(C_frag, 0.0f);

    int num_k_tiles = (K + WMMA_K - 1) / WMMA_K; 

    int threads_per_block = blockDim.x * blockDim.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    auto load_tile_to_shared = [&](int tile_idx, int buf_idx) {
        {
            const int A_elems = WMMA_M * WMMA_K;
            for (int i = tid; i < A_elems; i += threads_per_block) {
                int r = i / WMMA_K;
                int c = i % WMMA_K;
                int global_r = (warp_m * WMMA_M) + r;
                int global_c = tile_idx * WMMA_K + c;

                A_tile[buf_idx][r][c] = (global_r < M && global_c < N) ? A[global_r * N + global_c] : __float2half(0.0f);
            }
        }
        {
            const int B_elems = WMMA_K * WMMA_N;
            for (int i = tid; i < B_elems; i += threads_per_block) {
                int r = i / WMMA_N;
                int c = i % WMMA_N;
                int global_r = tile_idx * WMMA_K + r;              
                int global_c = (warp_n * WMMA_N) + c;         

                B_tile[buf_idx][r][c] = (global_r < K && global_c < N) ? B[global_r * N + global_c] : __float2half(0.0f);
                
            }
        }
    };

    load_tile_to_shared(0, 0);
    if (num_k_tiles > 1) load_tile_to_shared(1, 1);
    __syncthreads();

    if (num_k_tiles > 0) {
        wmma::load_matrix_sync(A_frag[0], &A_tile[0][0][0], WMMA_K);
        wmma::load_matrix_sync(B_frag[0], &B_tile[0][0][0], WMMA_N);
    }
    if (num_k_tiles > 1) {
        wmma::load_matrix_sync(A_frag[1], &A_tile[1][0][0], WMMA_K);
        wmma::load_matrix_sync(B_frag[1], &B_tile[1][0][0], WMMA_N);
    }

    for (int i = 0; i < num_k_tiles; ++i) {
        int buf_i   = i % 3;
        int buf_i1  = (i + 1) % 3;
        int buf_i2  = (i + 2) % 3;

        if (i + 2 < num_k_tiles) {
            load_tile_to_shared(i + 2, buf_i2);
        }

        __syncthreads();

        if (i + 1 < num_k_tiles) {
            wmma::load_matrix_sync(A_frag[buf_i1], &A_tile[buf_i1][0][0], WMMA_K);
            wmma::load_matrix_sync(B_frag[buf_i1], &B_tile[buf_i1][0][0], WMMA_N);
        }

        wmma::mma_sync(C_frag, A_frag[buf_i], B_frag[buf_i], C_frag);
    }

    // write results
    int out_row = warp_m * WMMA_M;
    int out_col = warp_n * WMMA_N;

    float c_tmp[WMMA_M * WMMA_N];
    wmma::store_matrix_sync(c_tmp, C_frag, WMMA_N, wmma::mem_row_major);

    for (int r = 0; r < WMMA_M; ++r) {
        int global_r = out_row + r;
        if (global_r >= M) continue;
        for (int c = 0; c < WMMA_N; ++c) {
            int global_c = out_col + c;
            if (global_c >= N) continue;
            C[ global_r * N + global_c ] = __float2half( c_tmp[r * WMMA_N + c]);
        }
    }
}


// Assumes WMMA_M = WMMA_N = WMMA_K = 16
__device__ float warpReduceMax(float val, unsigned mask = 0xffffffffu) {
    for (int offset = 16; offset > 0; offset /= 2)
        val = fmaxf(val, __shfl_down_sync(mask, val, offset));
    return val;
}
__device__ float warpReduceSum(float val, unsigned mask = 0xffffffffu) {
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(mask, val, offset);
    return val;
}

__global__ void fwd_attention(const half* __restrict__ d_Q,
                          const half* __restrict__ d_K_T,
                          const half* __restrict__ d_V,
                          half* __restrict__ d_out,
                          const int N,    // sequence length
                          const int d)    // head dimension
{
    // thread / block coords
    const int local_row = threadIdx.y;           // 0..WMMA_M-1
    const int local_col = threadIdx.x;           // 0..31 (warp lane)
    const int threads_per_block = blockDim.x * blockDim.y;
    const int tid = local_row * blockDim.x + local_col;

    const int tile_row_idx = blockIdx.y; // which Q-row tile (along N)
    const int tile_col_idx = blockIdx.x; // which output-column tile (along d)

    const int num_k_tiles = (d + WMMA_K - 1) / WMMA_K;

    // REQUIRE: one warp per WMMA row mapping (we use lane shuffles per row)
    if (!(blockDim.x == 32 && blockDim.y == WMMA_M)) {
        if (tid == 0) {
            printf("fwd_attention kernel requires blockDim.x==32 and blockDim.y==WMMA_M (WMMA_M=%d, blockDim.y=%d)\\n", WMMA_M, blockDim.y);
        }
        return;
    }

    // Shared memory tiles & buffers
    __shared__ half Q_tile[2][WMMA_M][WMMA_K];        // M x K  (row-major)
    __shared__ half Kt_tile[2][WMMA_K][WMMA_N];      // K x N  (row-major)
    __shared__ half V_tile[2][WMMA_N][WMMA_K];       // N x K  (row-major)

    // keep float intermediates for accuracy
    __shared__ float qkt_shared[2][WMMA_M][WMMA_N];  // M x N (float)
    __shared__ float softmax_out[2][WMMA_M][WMMA_N]; // numerators as float

    // half buffer for WMMA load
    __shared__ half softmax_half_buf[2][WMMA_M][WMMA_N];

    __shared__ float row_max[WMMA_M];
    __shared__ float row_den[WMMA_M];
    // rescale factor to apply to previous numerator O_old when max changes:
    __shared__ float rescale_factors[WMMA_M];

    // WMMA fragments (row-major layout)
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_K, WMMA_N, half, wmma::row_major> sm_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_K, WMMA_N, half, wmma::row_major> v_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_K, WMMA_N, float> final_c_frag;

    constexpr int elements_per_cp = 8;   // 8 half elements == 16 bytes
    constexpr int bytes_per_cp = 16;

    // ---------- loaders (fixed indices and bounds) ----------
    auto load_q_tile = [&](int k_tile_idx, int buf) {
        const int flat = WMMA_M * WMMA_K;
        const half* gmem_dst_base = d_Q;
        half* smem_dst = &Q_tile[buf][0][0];

        for (int i = tid * elements_per_cp; i < flat; i += threads_per_block * elements_per_cp) {
            int r = i / WMMA_K;            // 0..WMMA_M-1
            int c = i % WMMA_K;            // 0..WMMA_K-1
            int global_row = tile_row_idx * WMMA_M + r;  // along N
            int global_col = k_tile_idx * WMMA_K + c;    // along d

            const half* gmem_ptr = gmem_dst_base + global_row * d + global_col;
            half* smem_ptr = smem_dst + r * WMMA_K + c;

            if (global_row < N && global_col < d) {
                asm volatile("cp.async.ca.shared.global [%0], [%1], %2;\n" ::
                             "r"(smem_ptr), "l"(gmem_ptr), "n"(bytes_per_cp));
            }
        }
    };

    // K^T tile: d_K_T is (d x N) row-major; we want WMMA_K x WMMA_N tile
    auto load_kt_tile = [&](int k_tile_idx, int buf) {
        const int flat = WMMA_K * WMMA_N;
        const half* gmem_dst_base = d_K_T;
        half* smem_dst = &Kt_tile[buf][0][0];

        for (int i = tid * elements_per_cp; i < flat; i += threads_per_block * elements_per_cp) {
            int r = i / WMMA_N;            // 0..WMMA_K-1
            int c = i % WMMA_N;            // 0..WMMA_N-1
            int global_row = k_tile_idx * WMMA_K + r;          // along d
            int global_col = tile_row_idx * WMMA_N + c;        // along N

            const half* gmem_ptr = gmem_dst_base + global_row * N + global_col; // d_K_T row-major
            half* smem_ptr = smem_dst + r * WMMA_N + c;

            if (global_row < d && global_col < N) {
                asm volatile("cp.async.ca.shared.global [%0], [%1], %2;\n" ::
                             "r"(smem_ptr), "l"(gmem_ptr), "n"(bytes_per_cp));
            }
        }
    };

    // V tile: d_V is (N x d) row-major. We want WMMA_N x WMMA_K tile:
    auto load_v_tile = [&](int k_tile_idx, int buf) {
        const int flat = WMMA_N * WMMA_K;
        const half* gmem_dst_base = d_V;
        half* smem_dst = &V_tile[buf][0][0];

        for (int i = tid * elements_per_cp; i < flat; i += elements_per_cp * threads_per_block) {
            int r = i / WMMA_K;            // 0..WMMA_N-1
            int c = i % WMMA_K;            // 0..WMMA_K-1
            int global_row = tile_row_idx * WMMA_N + r;      // along N
            int global_col = k_tile_idx * WMMA_K + c;        // along d

            const half* gmem_ptr = gmem_dst_base + global_row * d + global_col;
            half* smem_ptr = smem_dst + r * WMMA_K + c;

            if (global_row < N && global_col < d) {
                asm volatile("cp.async.ca.shared.global [%0], [%1], %2;\n" ::
                             "r"(smem_ptr), "l"(gmem_ptr), "n"(bytes_per_cp));
            }
        }
    };

    // PREP: Preloading into first buffer (if any tiles)
    if (num_k_tiles > 0) {
        load_q_tile(0, 0);
        load_kt_tile(0, 0);
        load_v_tile(0, 0);
    }
    asm volatile("cp.async.commit_group;");

    // initialize accumulators and running-softmax stats
    wmma::fill_fragment(final_c_frag, 0.0f);

    if (local_col == 0 && local_row < WMMA_M) {
        row_max[local_row] = -1e30f;
        row_den[local_row] = 0.0f;
        rescale_factors[local_row] = 1.0f; // neutral initial scale
    }
    __syncthreads();

    // Precompute an active-lane mask that covers lanes [0..WMMA_N-1]
    unsigned full_mask = __activemask();
    unsigned active_mask;
    if (WMMA_N == 32) active_mask = full_mask;
    else active_mask = ( (WMMA_N >= 32) ? 0xffffffffu : ((1u << WMMA_N) - 1u) );

    // iterate k-tiles
    for (int k_tile_idx = 0; k_tile_idx < num_k_tiles; ++k_tile_idx) {
        int compute_buf = k_tile_idx & 1;
        int next_k = k_tile_idx + 1;

        // start loading next tile into the other buffer (if exists)
        if (next_k < num_k_tiles) {
            int load_buf = next_k & 1;
            load_q_tile(next_k, load_buf);
            load_kt_tile(next_k, load_buf);
            load_v_tile(next_k, load_buf);
        }

        asm volatile("cp.async.commit_group;");
        asm volatile("cp.async.wait_group 1;");

        // ---------------------
        // Q * K^T -> c_frag (M x N)
        // ---------------------
        wmma::fill_fragment(c_frag, 0.0f);
        wmma::load_matrix_sync(a_frag, &Q_tile[compute_buf][0][0], WMMA_K);   // ld = WMMA_K (Q row length)
        wmma::load_matrix_sync(b_frag, &Kt_tile[compute_buf][0][0], WMMA_N);  // ld = WMMA_N (Kt row length)
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

        const float scale = 1.0f / sqrtf((float)d);
        for (int i = 0; i < c_frag.num_elements; ++i) c_frag.x[i] *= scale;

        // store partial (M x N) into float qkt_shared
        wmma::store_matrix_sync(&qkt_shared[compute_buf][0][0], c_frag, WMMA_N, wmma::mem_row_major);
        __syncthreads();

        // ---------------------
        // Online Softmax (per-row, warp-parallel)
        // ---------------------
        {
            const int r = local_row; // row in WMMA_M (0..WMMA_M-1)
            const int lane = local_col; // lane id (0..31)

            // safe read: qkt_shared is M x N with N<=32; lanes >= WMMA_N will use -INFINITY
            float my_val = (r < WMMA_M && lane < WMMA_N) ? qkt_shared[compute_buf][r][lane] : -INFINITY;

            // each warp handles one row; lanes >= WMMA_N are inactive
            unsigned mask = active_mask; // only lanes 0..WMMA_N-1 participate

            // compute row max across active lanes
            float chunk_max = warpReduceMax(my_val, mask);

            // exponentiate relative to chunk_max
            float my_exp = (r < WMMA_M && lane < WMMA_N) ? expf(my_val - chunk_max) : 0.0f;

            // compute row sum across active lanes
            float chunk_sum = warpReduceSum(my_exp, mask);

            // lane 0 updates running stats and writes rescale_factors[row]
            float new_max_local = -INFINITY; // init for all lanes
            if ((lane & 0x1f) == 0 && r < WMMA_M) { // lane 0 within warp
                float prev_max = row_max[r];
                float prev_den = row_den[r];

                if (prev_den == 0.0f) {
                    // first tile for this row
                    row_max[r] = chunk_max;
                    row_den[r] = chunk_sum;
                    rescale_factors[r] = 0.0f;
                } else {
                    float new_max = fmaxf(prev_max, chunk_max);
                    float scaled_prev = expf(prev_max - new_max) * prev_den;
                    float scaled_chunk = expf(chunk_max - new_max) * chunk_sum;
                    rescale_factors[r] = expf(prev_max - new_max);
                    row_max[r] = new_max;
                    row_den[r] = scaled_prev + scaled_chunk;
                }
                new_max_local = row_max[r];
            }

            // broadcast updated max to all lanes in warp from lane 0 (works even if lane>0)
            float new_max = __shfl_sync(mask, new_max_local, 0);

            // normalized numerator (w.r.t. new_max)
            float my_num = (r < WMMA_M && lane < WMMA_N) ? expf(my_val - new_max) : 0.0f;
            if (r < WMMA_M && lane < WMMA_N) {
                softmax_out[compute_buf][r][lane] = my_num;
            }
        } // end online softmax

        // convert softmax_out (float) -> half buffer for WMMA
        {
            const int flat = WMMA_M * WMMA_N;
            for (int i = tid; i < flat; i += threads_per_block) {
                int r = i / WMMA_N;
                int c = i % WMMA_N;
                float v = softmax_out[compute_buf][r][c];
                softmax_half_buf[compute_buf][r][c] = __float2half(v);
            }
        }
        __syncthreads();

        // apply rescale to previous accumulator fragment
        for (int i = 0; i < final_c_frag.num_elements; ++i) {
            int rr = i / WMMA_K; // row within 0..WMMA_M-1
            float scale_factor = rescale_factors[rr];
            final_c_frag.x[i] *= scale_factor;
        }

        wmma::load_matrix_sync(sm_frag, &softmax_half_buf[compute_buf][0][0], WMMA_N); // ld = WMMA_N
        wmma::load_matrix_sync(v_frag,  &V_tile[compute_buf][0][0],           WMMA_K); // ld = WMMA_K
        wmma::mma_sync(final_c_frag, sm_frag, v_frag, final_c_frag);
        __syncthreads();
    } // end k_tile loop

    // write final result tile to global memory
    {
        float tmpC[WMMA_M * WMMA_K];
        wmma::store_matrix_sync(tmpC, final_c_frag, WMMA_K, wmma::mem_row_major); // ld = WMMA_K

        const int total_out_elems = WMMA_M * WMMA_K;
        for (int i = tid; i < total_out_elems; i += threads_per_block) {
            int rr = i / WMMA_K; // local row in tile
            int cc = i % WMMA_K; // local col in tile
            int global_r = tile_row_idx * WMMA_M + rr;
            int global_c = tile_col_idx * WMMA_K + cc;
            if (global_r < N && global_c < d) {
                float denom = row_den[rr];
                float val = tmpC[ rr * WMMA_K + cc ];
                float scaled = denom > 0.0f ? (val / denom) : 0.0f;
                d_out[ global_r * d + global_c ] = __float2half(scaled);
            }
        }
    }
}




    

__global__ void split_fused(const half* __restrict__ fused,
                            half* __restrict__ Q_out,
                            half* __restrict__ K_out,
                            half* __restrict__ V_out,
                            int seq_len, int Dk, int Dv)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int Nf = 2 * Dk + Dv;
    if (row >= seq_len || col >= (Dk > Dv ? Dk : Dv)) return; 

    if (row < seq_len) {
        int base = row * Nf;
        if (col < Dk) {
            Q_out[row * Dk + col] = fused[base + col];
        }
        if (col < Dk) {
            K_out[col * seq_len + row] = fused[base + Dk + col]; // Transpose K here (most opportune time)
        }
        if (col < Dv) {
            V_out[row * Dv + col] = fused[base + 2 * Dk + col];
        }
    }
}


int main() {

    const int M = SEQ_LEN;
    const int K_dim = D_MODEL;
    const int Dk = D_K;
    const int Dv = D_V;
    const int NFUSED = 2 * Dk + Dv;

    hipStream_t s0;
    CUDA_CHECK(hipStreamCreate(&s0));

    // Allocate ALL device memory needed for the graph beforehand
    half *d_X, *d_fusedW, *d_fused_out, *d_Q, *d_K, *d_V, *d_out;
    CUDA_CHECK(hipMalloc((void**)&d_X, sizeof(half) * M * K_dim));
    CUDA_CHECK(hipMalloc((void**)&d_fusedW, sizeof(half) * K_dim * NFUSED));
    CUDA_CHECK(hipMalloc((void**)&d_fused_out, sizeof(half) * M * NFUSED));
    CUDA_CHECK(hipMalloc((void**)&d_Q, sizeof(half) * M * Dk));
    CUDA_CHECK(hipMalloc((void**)&d_K, sizeof(half) * M * Dk));
    CUDA_CHECK(hipMalloc((void**)&d_V, sizeof(half) * M * Dv));
    CUDA_CHECK(hipMalloc((void**)&d_out, sizeof(half) * M * Dv));

    dim3 tpb(16,16);
    dim3 bpg_X( CEIL_DIV(K_dim, tpb.x), CEIL_DIV(M, tpb.y) );
    init_random_matrix<<<bpg_X, tpb, 0, s0>>>(d_X, M, K_dim, (unsigned long)time(NULL));

    dim3 bpg_fused_init( CEIL_DIV(NFUSED, tpb.x), CEIL_DIV(K_dim, tpb.y) );
    init_random_matrix<<<bpg_fused_init, tpb, 0, s0>>>(d_fusedW, K_dim, NFUSED, (unsigned long)time(NULL)+42);

    CUDA_CHECK(hipStreamSynchronize(s0));

    hipGraph_t graph;
    

    CUDA_CHECK(hipStreamBeginCapture(s0, hipStreamCaptureModeGlobal));


    dim3 mm_tpb(TILE_WIDTH, TILE_WIDTH);
    dim3 mm_bpg( CEIL_DIV(NFUSED, mm_tpb.x), CEIL_DIV(M, mm_tpb.y) );
    matmul<<<mm_bpg, mm_tpb, 0, s0>>>(d_X, d_fusedW, d_fused_out, M, NFUSED, K_dim);

    dim3 split_tpb(16, 16);
    dim3 split_bpg( CEIL_DIV(NFUSED, split_tpb.x), CEIL_DIV(M, split_tpb.y) );
    split_fused<<<split_bpg, split_tpb, 0, s0>>>(d_fused_out, d_Q, d_K, d_V, M, Dk, Dv);

    dim3 attn_tpb(32, WMMA_M);
    dim3 attn_bpg(CEIL_DIV(Dv, WMMA_K), CEIL_DIV(M, WMMA_M));
    fwd_attention<<<attn_bpg, attn_tpb, 0, s0>>>(d_Q, d_K, d_V, d_out, M, Dv);
    
    CUDA_CHECK(hipStreamEndCapture(s0, &graph));


    hipGraphExec_t graph_exec;
    CUDA_CHECK(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));


    const int num_runs = 100;
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start, s0));
    for (int i = 0; i < num_runs; ++i) {
        CUDA_CHECK(hipGraphLaunch(graph_exec, s0));
    }
    CUDA_CHECK(hipEventRecord(stop, s0));
    CUDA_CHECK(hipStreamSynchronize(s0));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Execution time for %d graph launches: %.3f ms\n", num_runs, milliseconds);
    printf("Average time per launch: %.6f ms\n", milliseconds / num_runs);


    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_fusedW));
    CUDA_CHECK(hipFree(d_fused_out));
    CUDA_CHECK(hipFree(d_Q));
    CUDA_CHECK(hipFree(d_K));
    CUDA_CHECK(hipFree(d_V));
    CUDA_CHECK(hipFree(d_out));

    CUDA_CHECK(hipStreamDestroy(s0));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    
    CUDA_CHECK(hipGraphDestroy(graph));
    CUDA_CHECK(hipGraphExecDestroy(graph_exec));

    return 0;
}