#include "hip/hip_runtime.h"
/*
Very simple file just to implement a timer with events. Timing a small kernel.
*/
#include <hip/hip_runtime.h>
#include <studio.h>

#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, cudaGetStringError(err)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)


__global__ void Kernel(float* input, float* output, int N) {
    if (threadIdx.x < N) {
        output[threadIdx.x] = input[threadIdx.x] * 5.0f;
    }
    
}

int main() {
    int N = 5000;
    float* h_inp, h_out;
    size_t size = N * sizeof(float);


    CUDA_CHECK(hipHostMalloc((void**)&h_inp, size));
    CUDA_CHECK(hipHostMalloc((void**)&h_out, size));

    for (int i = 0; i < N; ++i) { inp[i] = i; }

    float* d_inp, d_out;

    CUDA_CHECK(hipMalloc((void**)&d_inp, size));
    CUDA_CHECK(hipMalloc((void**)&d_out, size));

    CUDA_CHECK(hipMemcpy(d_inp, h_inp, size, hipMemcpyHostToDevice));

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);


    dim3 threadsPerBlock(N);

    CUDA_CHECK(hipEventRecord(start));

    Kernel<<<1, N>>>(inp, out, N);

    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Kernel execution time: %f ms\n", milliseconds);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(cudaEventDestory(stop));

    CUDA_CHECK(hipFree(d_inp));
    CUDA_CHECK(hipFree(d_out));

    CUDA_CHECK(hipHostFree(h_inp));
    CUDA_CHECK(hipHostFree(h_out));

    return 0;


}