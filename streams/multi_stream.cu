#include "hip/hip_runtime.h"
/*
Stream Practice

Run Kernels A and B simultaneously, and C to wait on both
*/
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) \

do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "Cuda Error in file %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)


__global__ void kernelA(float* input, float* output, int N) {
    if (threadIdx.x < N) {
        output[threadIdx.x] = input[threadIdx.x] * 5.0f;
    }
}
__global__ void kernelB(float* input, float* output, int N) {
    if (threadIdx.x < N) {
        output[threadIdx.x] = input[threadIdx.x] * 4.0f;
    }
}
__global__ void kernelC(float* input, float* output, int N) {
    if (threadIdx.x < N) {
        output[threadIdx.x] = input[threadIdx.x] * 10.0f;
    }
}

int main() {
    float *h_inpA, *h_outA;
    int N = 1000;

    size_t size = N * sizeof(float);

    CUDA_CHECK(hipHostMalloc((void**)&h_inpA, size));
    CUDA_CHECK(hipHostMalloc((void**)&h_outA, size));

    for (int i = 0; i < N; ++i) { h_inpA[i] = i % 15; }

    float *h_inpB, *h_outB;

    CUDA_CHECK(hipHostMalloc((void**)&h_inpB, size));
    CUDA_CHECK(hipHostMalloc((void**)&h_outB, size));


    float *d_inpA, *d_outA;

    CUDA_CHECK(hipMalloc((void**)&d_inpA, size));
    CUDA_CHECK(hipMalloc((void**)&d_outA, size));

    for (int i = 0; i < N; ++i) { h_inpA[i] = i % 15; }

    float *d_inpB, *d_outB;

    CUDA_CHECK(hipMalloc((void**)&d_inpB, size));
    CUDA_CHECK(hipMalloc((void**)&d_outB, size));

    for (int i = 0; i < N; ++i) { h_inpB[i] = i % 20; }

    float *h_inpC, *h_outC;

    CUDA_CHECK(hipHostMalloc((void**)&h_inpC, size));
    CUDA_CHECK(hipHostMalloc((void**)&h_outC, size));
    

    float *d_inpC, *d_outC;

    CUDA_CHECK(hipMalloc((void**)&d_inpC, size));
    CUDA_CHECK(hipMalloc((void**)&d_outC, size));


    hipEvent_t memA, memB, kernel_completion;
    
    hipEventCreate(&memA);
    hipEventCreate(&memB);
    hipEventCreate(&kernel_completion);


    hipStream_t s1, s2;

    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    CUDA_CHECK(cudaMemcpyAsynch())



}