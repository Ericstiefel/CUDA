#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <math.h>

#define VOCAB_LEN 1000

#define CUDA_CHECK(call)
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error in file %s in line %d: %s", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// Necessary to init random weight matrices & feedforward layers
__global__ void init_random_matrix(half* ptr, int rows, int cols, unsigned long seed) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < rows && col < cols) {
        int idx = row * cols + col;
        hiprandState localState;
        curandInit(seed, idx, 0, &localState);
        float rv = hiprand_uniform(&localState);
        ptr[idx] = __float2half(rv);
    }
}

/*
Place the inp tokens into constant memory, O(5) CS access, and len(sequence) * sizeof(int) will fit into constant memory using any reasonable sequence length

dim3 threadsPerBlock(M)
dim3 blocksPerGrid(N)

Each block will embed a single token.

max threadsPerBlock -> 1024, assume token input length is less than that.
max blocksPerGrid -> 2^31 - 1, we are substantially below.
*/

__global__ void encodings(half* __restrict__ inp, half* __restrict__ out, int M, int N){ // Inp is a vector of length N. Each token will be encoded by a vector of length M, generating a M x N out matrix.
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    /*
    Simple encoding function (obviously learned in a real LLM)

    sin(r * (vocab_idx)) (yes I know there will be recurring values because trig functions oscillate and reach the same values every sin(2x). This is just purely for demonstration.
    */
   
    if (row < M && col < N) {
        int idx = row * N + col;

        float comp = row * inp[row];

        out[idx] = __float2half(__sinf(__float2half(comp)));
    }

}

/*

dim3 threadsPerBlock(M);
dim3 blocksPerGrid(N);

Launched in this way to ensure all threads in a warp (assumed by them being in the same block) don't diverge with trig split causing thread divergence, a significant slowdown.
*/

__global__ void positional_encodings(
    const half* __restrict__ inp, 
    half* __restrict__ out,
    int N, 
    int M    
) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // position index (pos)
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // embedding dimension (dim)

    // No need for check, if launched with simple dims as above

    int idx = row * M + col; 

    float pos = static_cast<float>(row);
    float i = static_cast<float>(col / 2);  
    float denom = powf(10000.0f, (2.0f * i) / static_cast<float>(M));

    float angle = pos / denom;

    float val = (col % 2 == 0) ? __sinf(angle) : __cosf(angle);

    out[idx] = __float2half(__half2float(inp[idx]) + val);
}
